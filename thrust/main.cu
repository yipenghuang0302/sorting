#include "hip/hip_runtime.h"
#include "hamiltonian.h"
#include <thrust/sort.h>

template <typename T> struct square {
	__host__ __device__
	T operator()(const T& x) const { 
		return x * x;
	}
};

double rms ( size_t n, device_type a, device_type b ) {
	double sum = 0.0;
	for ( size_t idx=0; idx<n; idx++ )
		sum += (a[idx]-b[idx]) * (a[idx]-b[idx]);
	return sqrt(sum/n);
}

int main( int argc , char **argv ) {

	srand (time(NULL));

	// dynamic range of real numbers we are sorting
	for (double dt=1.0/256.0; 1.0/256.0<=dt; dt=dt/2.0) {
		// range of off-diagonals
		for (double range=2.0; range<=2.0; range*=2.0) {
			for (double interval=16.0; interval<=16.0; interval*=2.0) {

				// number of reals to sort
				// we pad with two boundary elements at front and back
				for (size_t n=16; n<=16; n*=2) {
					for (size_t trial=0; trial<=0; trial++) {

						// q is position in Hamiltonian system; in sorting system they are related to off-diagonals
						// p is momentum in Hamiltonian system; in sorting system they are related to the set to sort
						device_type q( n , 0.0 ) , p_sort( n , 0.0 ), p_curr( n , 0.0 ), p_prev( n, 0.0 );
						q[0] = -HUGE_VAL; // q_0 is initialized to negative infinity
						for( size_t i=1 ; i<n-1 ; i++ )
						{
							q[i] = i*interval; // we choose this to ensure that the off-diagonals are small
							p_curr[i] = p_sort[i] = -range + (float)(rand()) / (float)(RAND_MAX/(range-(-range)));
						}
						q[n-1] = HUGE_VAL; // q_0 is initialized to positive infinity
						thrust::sort (p_sort.begin()+1, p_sort.end()-1);

						// copy to GPU side
						// device_type p_device = p_host;
						// device_type q_device = q_host;

						thrust::copy( p_curr.begin() , p_curr.end() , std::ostream_iterator< double >( std::cout , ", " ) );
						std::cout << std::endl;
						thrust::copy( p_sort.begin() , p_sort.end() , std::ostream_iterator< double >( std::cout , ", " ) );
						std::cout << std::endl;

						stepper_type stepper;
						double error = DBL_MAX;
						double time = 0.0;
						do {
							p_prev = p_curr;
							stepper.do_step (
								momentum(n),
								std::make_pair( boost::ref( q ) , boost::ref( p_curr ) ) ,
								time,
								dt  // time step
								// streaming_observer( std::cout , 16*16*16 )
							);
							time += dt;
							thrust::transform (
								p_curr.begin(),
								p_curr.end(),
								p_prev.begin(),
								p_prev.begin(),
								thrust::minus<double>()
							);
							square<float> unary_op;
							thrust::plus<double> binary_op;
							error = std::sqrt(
								thrust::transform_reduce(
									p_prev.begin(),
									p_prev.end(),
									unary_op,
									0.0,
									binary_op
								)/n
							);
							printf("error =\t%e\n", error);
						} while ( DBL_EPSILON<error );

						double quality = rms(n, p_sort, p_curr);

						thrust::copy( p_curr.begin() , p_curr.end() , std::ostream_iterator< double >( std::cout , ", " ) );
						std::cout << std::endl;

						printf("dt =\t%e\trange =\t%e\tinterval =\t%e\tn =\t%ld\ttime =\t%e\tquality = \t%e\n", dt, range, interval, n, time, quality);
					}
				}
			}
		}
	}	
	return 0;
}